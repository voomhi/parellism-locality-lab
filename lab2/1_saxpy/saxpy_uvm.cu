#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

// static inline
// int getBlocks(long working_set_size, int threadsPerBlock) {
//   // TODO: implement and use this interface if necessary
//     return 0;
// }

void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
  // TODO: implement and use this interface if necessary
    hipMallocManaged(xarray, sizeof(float)*size);
    hipMallocManaged(yarray, sizeof(float)*size);
    hipMallocManaged(resultarray, sizeof(float)*size);

}

void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
  // TODO: implement and use this interface if necessary  
    hipFree(xarray);
    hipFree(yarray);
    hipFree(resultarray);

}

void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 32; // change this if necessary

    // float *device_x;
    // float *device_y;
    // float *device_result;

    //
    // TODO: do we need to allocate device memory buffers on the GPU here?
    // Yes can't do it in main due to not being able to include the files

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

  
    //
    // TODO: do we need copy here?
    // No
     
    //
    // TODO: insert time here to begin timing only the kernel
    //
    
    // compute number of blocks and threads per block
    double startGPUTime = CycleTimer::currentSeconds();

    // run saxpy_kernel on the GPU
    // printf("kernal0");
    int blocks = (total_elems + threadsPerBlock-1)/threadsPerBlock;
    saxpy_kernel<<<blocks,threadsPerBlock>>>(total_elems*sizeof(float),alpha,xarray,yarray,resultarray);


    //
    // TODO: insert timer here to time only the kernel.  Since the
    // kernel will run asynchronously with the calling CPU thread, you
    // need to call hipDeviceSynchronize() before your timer to
    // ensure the kernel running on the GPU has completed.  (Otherwise
    // you will incorrectly observe that almost no time elapses!)
    //
    hipDeviceSynchronize();
    double endGPUTime = CycleTimer::currentSeconds();
    double timeKernel = endGPUTime - startGPUTime;
    
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
    
    //
    // TODO: copy result from GPU using hipMemcpy
    //

    // What would be copy time when we use UVM?
    //Kernal runTime
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    totalTimeAvg   += overallDuration;
    timeKernelAvg += timeKernel;
    //
    // TODO free device memory if you allocate some device memory earlier in this function.
    //
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
